#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

#include "dropout_layer.h"
#include "hip/hip_runtime.h"
#include "utils.h"

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id < size) input[id] = (rand[id] < prob) ? 0 : input[id]*scale;
}

void forward_dropout_layer_gpu(dropout_layer l, network net)
{
    if (!net.train) return;
    int size = l.inputs*l.batch;
    cuda_random(l.rand_gpu, size);
    /*
    int i;
    for(i = 0; i < size; ++i){
        l.rand[i] = rand_uniform();
    }
    cuda_push_array(layer.rand_gpu, l.rand, size);
    */

    yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK>>>(net.input_gpu, size, l.rand_gpu, l.probability, l.scale);
    check_error(hipPeekAtLastError());
}

void backward_dropout_layer_gpu(dropout_layer l, network net)
{
    if(!net.delta_gpu) return;
    int size = l.inputs*l.batch;

    yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK>>>(net.delta_gpu, size, l.rand_gpu, l.probability, l.scale);
    check_error(hipPeekAtLastError());
}
